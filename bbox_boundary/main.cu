#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/transform_reduce.h>

#include "timer.hpp"

// This example shows how to compute a bounding box
// for a set of points in two dimensions.

struct point2d {
    float x, y;

    __host__ __device__ point2d() : x(0), y(0) {}

    __host__ __device__ point2d(float _x, float _y) : x(_x), y(_y) {}
};

// bounding box type
struct bbox {
    // construct an empty box
    __host__ __device__ bbox() {}

    // construct a box from a single point
    __host__ __device__ bbox(const point2d &point)
        : lower_left(point), upper_right(point) {}

    // construct a box from a single point
    __host__ __device__ bbox &operator=(const point2d &point) {
        lower_left = point;
        upper_right = point;
        return *this;
    }

    // construct a box from a pair of points
    __host__ __device__ bbox(const point2d &ll, const point2d &ur)
        : lower_left(ll), upper_right(ur) {}

    point2d lower_left, upper_right;
};

// reduce a pair of bounding boxes (a,b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox, bbox, bbox> {
    __host__ __device__ bbox operator()(bbox a, bbox b) {
        // lower left corner
        point2d ll(thrust::min(a.lower_left.x, b.lower_left.x),
                   thrust::min(a.lower_left.y, b.lower_left.y));

        // upper right corner
        point2d ur(thrust::max(a.upper_right.x, b.upper_right.x),
                   thrust::max(a.upper_right.y, b.upper_right.y));

        return bbox(ll, ur);
    }
};

/* results
const size_t N = 1'0000'0000;
    GPU
        t_main - Execution time: 2.462 milli seconds
        bounding box (0.000000,0.000000) (1.000000,1.000000)

    CPU
        t_main - Execution time: 222.203 milli seconds
        bounding box (0.000000,0.000000) (1.000000,1.000000)

const size_t N = 1000'0000;
    GPU
        t_main - Execution time: 0.432 milli seconds
        bounding box (0.000000,0.000000) (1.000000,1.000000)

    CPU
        t_main - Execution time: 22.209 milli seconds
        bounding box (0.000000,0.000000) (1.000000,1.000000)
*/
int main(void) {
    const size_t N = 1'0000'0000;

    thrust::default_random_engine rng;
    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

    // allocate storage for points
    std::vector<point2d> host_points(N);
    thrust::device_vector<point2d> points(N);

    // generate some random points in the unit square on the host
    for (size_t i = 0; i < N; i++) {
        float x = u01(rng);
        float y = u01(rng);
        host_points[i] = point2d(x, y);
    }

    // copy points from host to device
    thrust::copy(host_points.begin(), host_points.end(), points.begin());

    // initial bounding box contains first point
    bbox init = bbox(points[0], points[0]);

    // binary reduction operation
    bbox_reduction binary_op;

    // compute the bounding box for the point set
    bbox result{};
    double t_main = tictoc(
        [&]() {
            return thrust::reduce(points.begin(), points.end(), init,
                                  binary_op);
        },
        result);

    print_time(t_main, "t_main");

    // print output
    std::cout << "bounding box " << std::fixed;
    std::cout << "(" << result.lower_left.x << "," << result.lower_left.y
              << ") ";
    std::cout << "(" << result.upper_right.x << "," << result.upper_right.y
              << ")" << std::endl;

    return 0;
}