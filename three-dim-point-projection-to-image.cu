#include "hip/hip_runtime.h"
/*
    try test yourself!
    https://godbolt.org/z/x4G73af9a
*/

#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <iostream>
#include <cmath>
#include <thrust/copy.h>
#include <hip/hip_runtime.h>
#include <vector>

// 구조체를 정의하여 3D 포인트를 구면 좌표로 변환
struct ToSpherical {
    const float width, height;

    ToSpherical(float w, float h) : width(w), height(h) {}

    __device__ float2 operator()(float3 point) {
        float radius = sqrtf(point.x * point.x + point.y * point.y + point.z * point.z);
        float azimuth = atan2f(point.y, point.x);
        float elevation = acosf(point.z / radius);

        // 방위각과 고도를 이미지 좌표로 변환
        float u = (azimuth + M_PI) / (2 * M_PI) * width;
        float v = elevation / M_PI * height;
        return make_float2(u, v);
    }
};

// 무작위 포인트 생성을 위한 구조체
struct RandomPointGenerator {
    unsigned int seed;

    RandomPointGenerator(unsigned int s) : seed(s) {}

    __device__ float3 operator()(const unsigned int n) const {
        thrust::default_random_engine rng(seed);
        thrust::uniform_real_distribution<float> dist(-10.0, 10.0);
        rng.discard(n);

        return make_float3(dist(rng), dist(rng), dist(rng));
    }
};

void printMemoryUsage(const char* stage) {
    size_t free_byte;
    size_t total_byte;
    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

    if (hipSuccess != cuda_status){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
        exit(1);
    }

    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    printf("Memory usage at %s: used = %f, free = %f MB, total = %f MB\n",
            stage, used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}


int main() {

    // GPU 속성 가져오기
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    std::cout << "Using GPU: " << properties.name << std::endl;

    // CUDA 버전 가져오기
    int runtimeVer;
    hipRuntimeGetVersion(&runtimeVer);
    std::cout << "CUDA Runtime Version: " << runtimeVer / 1000 << "." << (runtimeVer % 100) / 10 << std::endl;

    int driverVer;
    hipDriverGetVersion(&driverVer);
    std::cout << "CUDA Driver Version: " << driverVer / 1000 << "." << (driverVer % 100) / 10 << std::endl;

    // 
    const size_t N = 5000000;
    const int width = 640, height = 480;
    hipEvent_t start, stop;
    float totalTime = 0, elapsedTime = 0;

    // 이벤트 생성
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printMemoryUsage("before allocation");

    // 무작위 포인트 클라우드 생성
    thrust::device_vector<float3> d_points(N);

    hipEventRecord(start);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      d_points.begin(),
                      RandomPointGenerator(time(nullptr)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Point generation time: " << elapsedTime << " ms\n";
    totalTime += elapsedTime;

    // 포인트 클라우드를 구면 좌표로 변환
    thrust::device_vector<float2> d_image_coords(N);

    hipEventRecord(start);
    thrust::transform(d_points.begin(), d_points.end(), d_image_coords.begin(), ToSpherical(width, height));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Spherical transformation time: " << elapsedTime << " ms\n";
    totalTime += elapsedTime;

    printMemoryUsage("after allocation");

    // 결과를 호스트로 복사
    std::vector<float3> h_points(N);
    std::vector<float2> h_image_coords(N);

    hipEventRecord(start);
    thrust::copy(d_points.begin(), d_points.end(), h_points.begin());
    thrust::copy(d_image_coords.begin(), d_image_coords.end(), h_image_coords.begin());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Copying to host time: " << elapsedTime << " ms\n";
    totalTime += elapsedTime;

    // 호스트에서 결과 출력 (이 부분은 시간 측정 대상에서 제외)
    for (size_t i = 0; i < 100; i++) {
        std::cout << "Point " << i << ": (" << h_points[i].x << ", " << h_points[i].y << ", " << h_points[i].z << ")"
                  << " -> Image Coords: (" << h_image_coords[i].x << ", " << h_image_coords[i].y << ")" << std::endl;
    }

    // 전체 수행 시간 출력
    std::cout << "Total execution time: " << totalTime << " ms\n";

    // 이벤트 파괴
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}


/*
ASM generation compiler returned: 0
Execution build compiler returned: 0
Program returned: 0

Using GPU: Tesla T4
CUDA Runtime Version: 12.2
CUDA Driver Version: 12.2

ASM generation compiler returned: 0
Execution build compiler returned: 0
Program returned: 0
Memory usage at before allocation: used = 105.000000, free = 14825.562500 MB, total = 14930.562500 MB
Point generation time: 1.88826 ms
Spherical transformation time: 1.83283 ms
Memory usage at after allocation: used = 205.000000, free = 14725.562500 MB, total = 14930.562500 MB
Copying to host time: 473.374 ms
Point 0: (1.54322, -7.14025, -6.97402) -> Image Coords: (181.681, 356.458)
Point 1: (-7.14025, -6.97402, -3.01801) -> Image Coords: (78.8004, 284.864)
Point 2: (-6.97402, -3.01801, -2.43688) -> Image Coords: (41.601, 287.414)
Point 3: (-3.01801, -2.43688, 9.51612) -> Image Coords: (69.1892, 59.1389)
Point 4: (-2.43688, 9.51612, -7.5907) -> Image Coords: (505.535, 340.519)
Point 5: (9.51612, -7.5907, 9.08719) -> Image Coords: (251.416, 142.021)
Point 6: (-7.5907, 9.08719, 7.63673) -> Image Coords: (550.885, 152.478)
Point 7: (9.08719, 7.63673, -7.61795) -> Image Coords: (391.188, 327.178)
Point 8: (7.63673, -7.61795, -5.92422) -> Image Coords: (240.125, 316.737)
Point 9: (-7.61795, -5.92422, -7.91345) -> Image Coords: (67.3262, 344.939)
Point 10: (-5.92422, -7.91345, 9.83294) -> Image Coords: (94.5431, 120.406)
Point 11: (-7.91345, 9.83294, 5.82871) -> Image Coords: (549.025, 173.9)
Point 12: (9.83294, 5.82871, -2.1359) -> Image Coords: (374.504, 268.224)
Point 13: (5.82871, -2.1359, -1.89917) -> Image Coords: (284.222, 285.362)
Point 14: (-2.1359, -1.89917, 5.371) -> Image Coords: (74.0309, 74.7179)
Point 15: (-1.89917, 5.371, 3.77571) -> Image Coords: (514.619, 150.573)
Point 16: (5.371, 3.77571, -2.52031) -> Image Coords: (382.412, 296.002)
Point 17: (3.77571, -2.52031, 2.00765) -> Image Coords: (260.047, 176.38)

*/

/* 설명 포인트
- cpu gpu 카피 비용이 대부분이다. 포인트 수 증가해도 Point generation time 과 Spherical transformation time 은 작음
*/
